#include "hip/hip_runtime.h"
/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "yrt-pet/operators/DeviceSynchronized.cuh"

#include "yrt-pet/datastruct/image/Image.hpp"
#include "yrt-pet/utils/GPUUtils.cuh"
#include "yrt-pet/utils/Globals.hpp"


namespace yrt
{
namespace util
{

GPULaunchParams3D initiateDeviceParameters(const ImageParams& params)
{
	GPULaunchParams3D launchParams;
	if (params.nz > 1)
	{
		const size_t threadsPerBlockDimImage =
		    globals::ThreadsPerBlockImg3d;
		const auto threadsPerBlockDimImage_float =
		    static_cast<float>(threadsPerBlockDimImage);
		const auto threadsPerBlockDimImage_uint =
		    static_cast<unsigned int>(threadsPerBlockDimImage);

		launchParams.gridSize = {
		    static_cast<unsigned int>(
		        std::ceil(params.nx / threadsPerBlockDimImage_float)),
		    static_cast<unsigned int>(
		        std::ceil(params.ny / threadsPerBlockDimImage_float)),
		    static_cast<unsigned int>(
		        std::ceil(params.nz / threadsPerBlockDimImage_float))};

		launchParams.blockSize = {threadsPerBlockDimImage_uint,
		                          threadsPerBlockDimImage_uint,
		                          threadsPerBlockDimImage_uint};
	}
	else
	{
		const size_t threadsPerBlockDimImage =
		    globals::ThreadsPerBlockImg2d;
		const auto threadsPerBlockDimImage_float =
		    static_cast<float>(threadsPerBlockDimImage);
		const auto threadsPerBlockDimImage_uint =
		    static_cast<unsigned int>(threadsPerBlockDimImage);

		launchParams.gridSize = {
		    static_cast<unsigned int>(
		        std::ceil(params.nx / threadsPerBlockDimImage_float)),
		    static_cast<unsigned int>(
		        std::ceil(params.ny / threadsPerBlockDimImage_float)),
		    1};

		launchParams.blockSize = {threadsPerBlockDimImage_uint,
		                          threadsPerBlockDimImage_uint, 1};
	}
	return launchParams;
}

GPULaunchParams initiateDeviceParameters(size_t batchSize)
{
	GPULaunchParams launchParams{};
	launchParams.gridSize = static_cast<unsigned int>(std::ceil(
	    batchSize / static_cast<float>(globals::ThreadsPerBlockData)));
	launchParams.blockSize = globals::ThreadsPerBlockData;
	return launchParams;
}

}  // namespace util

const hipStream_t* DeviceSynchronized::getMainStream() const
{
	return mp_mainStream;
}

const hipStream_t* DeviceSynchronized::getAuxStream() const
{
	return mp_auxStream;
}

CUScannerParams DeviceSynchronized::getCUScannerParams(const Scanner& scanner)
{
	CUScannerParams params;
	params.crystalSize_trans = scanner.crystalSize_trans;
	params.crystalSize_z = scanner.crystalSize_z;
	params.numDets = scanner.getNumDets();
	return params;
}

CUImageParams DeviceSynchronized::getCUImageParams(const ImageParams& imgParams)
{
	CUImageParams params;

	params.voxelNumber[0] = imgParams.nx;
	params.voxelNumber[1] = imgParams.ny;
	params.voxelNumber[2] = imgParams.nz;

	params.imgLength[0] = imgParams.length_x;
	params.imgLength[1] = imgParams.length_y;
	params.imgLength[2] = imgParams.length_z;

	params.voxelSize[0] = imgParams.vx;
	params.voxelSize[1] = imgParams.vy;
	params.voxelSize[2] = imgParams.vz;

	params.offset[0] = imgParams.off_x;
	params.offset[1] = imgParams.off_y;
	params.offset[2] = imgParams.off_z;

	params.fovRadius = imgParams.fovRadius;

	return params;
}

DeviceSynchronized::DeviceSynchronized(const hipStream_t* pp_mainStream,
                                       const hipStream_t* pp_auxStream)
{
	mp_mainStream = pp_mainStream;
	mp_auxStream = pp_auxStream;
}

}  // namespace yrt
