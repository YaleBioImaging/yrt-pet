#include "hip/hip_runtime.h"
/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "operators/OperatorProjectorDD_GPU.cuh"

#include "datastruct/image/Image.hpp"
#include "datastruct/image/ImageDevice.cuh"
#include "datastruct/projection/ProjectionDataDevice.cuh"
#include "operators/OperatorProjectorDD_GPUKernels.cuh"
#include "utils/Assert.hpp"
#include "utils/GPUUtils.cuh"

#if BUILD_PYBIND11
#include <pybind11/pybind11.h>
namespace py = pybind11;

void py_setup_operatorprojectordd_gpu(py::module& m)
{
	auto c = py::class_<OperatorProjectorDD_GPU, OperatorProjectorDevice>(
	    m, "OperatorProjectorDD_GPU");
	c.def(py::init<const OperatorProjectorParams&>(), py::arg("projParams"));
	c.def(
	    "applyA",
	    [](OperatorProjectorDD_GPU& self, const ImageDevice* img,
	       ProjectionData* proj) { self.applyA(img, proj); },
	    py::arg("img"), py::arg("proj"));
	c.def(
	    "applyA",
	    [](OperatorProjectorDD_GPU& self, const Image* img,
	       ProjectionData* proj) { self.applyA(img, proj); },
	    py::arg("img"), py::arg("proj"));
	c.def(
	    "applyA",
	    [](OperatorProjectorDD_GPU& self, const ImageDevice* img,
	       ProjectionDataDevice* proj) { self.applyA(img, proj); },
	    py::arg("img"), py::arg("proj"));
	c.def(
	    "applyA",
	    [](OperatorProjectorDD_GPU& self, const Image* img,
	       ProjectionDataDevice* proj) { self.applyA(img, proj); },
	    py::arg("img"), py::arg("proj"));

	c.def(
	    "applyAH",
	    [](OperatorProjectorDD_GPU& self, const ProjectionData* proj,
	       Image* img) { self.applyAH(proj, img); },
	    py::arg("proj"), py::arg("img"));
	c.def(
	    "applyAH",
	    [](OperatorProjectorDD_GPU& self, const ProjectionData* proj,
	       ImageDevice* img) { self.applyAH(proj, img); },
	    py::arg("proj"), py::arg("img"));
	c.def(
	    "applyAH",
	    [](OperatorProjectorDD_GPU& self, const ProjectionDataDevice* proj,
	       Image* img) { self.applyAH(proj, img); },
	    py::arg("proj"), py::arg("img"));
	c.def(
	    "applyAH",
	    [](OperatorProjectorDD_GPU& self, const ProjectionDataDevice* proj,
	       ImageDevice* img) { self.applyAH(proj, img); },
	    py::arg("proj"), py::arg("img"));
}
#endif

OperatorProjectorDD_GPU::OperatorProjectorDD_GPU(
    const OperatorProjectorParams& projParams, bool p_synchronized,
    const hipStream_t* mainStream, const hipStream_t* auxStream)
    : OperatorProjectorDevice(projParams, p_synchronized, mainStream, auxStream)
{
}

void OperatorProjectorDD_GPU::applyA(const Variable* in, Variable* out)
{
	auto* img_in_const = dynamic_cast<const ImageDevice*>(in);
	auto* dat_out = dynamic_cast<ProjectionDataDevice*>(out);

	// In case the user provided a host-side image
	std::unique_ptr<ImageDeviceOwned> deviceImg_out = nullptr;
	ImageDevice* img_in = nullptr;
	if (img_in_const == nullptr)
	{
		const auto* hostImg_in = dynamic_cast<const Image*>(in);
		ASSERT_MSG(
		    hostImg_in != nullptr,
		    "The image provided is not a ImageDevice nor a Image (host)");

		deviceImg_out = std::make_unique<ImageDeviceOwned>(
		    hostImg_in->getParams(), getAuxStream());
		deviceImg_out->allocate(true);
		deviceImg_out->transferToDeviceMemory(hostImg_in, true);

		// Use owned ImageDevice
		img_in = deviceImg_out.get();
	}
	else
	{
		img_in = const_cast<ImageDevice*>(img_in_const);
		ASSERT_MSG(img_in != nullptr, "ImageDevice is null. Cast failed");
	}

	// In case the user provided a Host-side ProjectionData
	bool isProjDataDeviceOwned = false;
	std::unique_ptr<ProjectionDataDeviceOwned> deviceDat_out = nullptr;
	ProjectionData* hostDat_out = nullptr;
	if (dat_out == nullptr)
	{
		hostDat_out = dynamic_cast<ProjectionData*>(out);
		ASSERT_MSG(hostDat_out != nullptr,
		           "The Projection Data provded is not a ProjectionDataDevice "
		           "nor a ProjectionData (host)");
		ASSERT_MSG(binIter != nullptr, "BinIterator undefined");

		std::vector<const BinIterator*> binIterators;
		binIterators.push_back(binIter);  // We project only one subset
		deviceDat_out = std::make_unique<ProjectionDataDeviceOwned>(
		    getScanner(), hostDat_out, binIterators);

		// Use owned ProjectionDataDevice
		dat_out = deviceDat_out.get();
		isProjDataDeviceOwned = true;
	}

	if (!isProjDataDeviceOwned)
	{
		std::cout << "Forward projecting current batch..." << std::endl;
		applyOnLoadedBatch<true>(dat_out, img_in);
	}
	else
	{
		// Iterate over all the batches of the current subset
		const size_t numBatches = dat_out->getBatchSetup(0).getNumBatches();
		const ImageParams& imgParams = img_in->getParams();
		for (size_t batchId = 0; batchId < numBatches; batchId++)
		{
			std::cout << "Loading batch " << batchId + 1 << "/" << numBatches
			          << "..." << std::endl;
			dat_out->loadEventLORs(0, batchId, imgParams, getAuxStream());
			deviceDat_out->allocateForProjValues(getAuxStream());
			dat_out->clearProjectionsDevice(getMainStream());
			std::cout << "Forward projecting batch..." << std::endl;
			applyOnLoadedBatch<true>(dat_out, img_in);
			std::cout << "Transferring batch to Host..." << std::endl;
			dat_out->transferProjValuesToHost(hostDat_out, getAuxStream());
		}
	}
}

void OperatorProjectorDD_GPU::applyAH(const Variable* in, Variable* out)
{
	auto* dat_in_const = dynamic_cast<const ProjectionDataDevice*>(in);
	auto* img_out = dynamic_cast<ImageDevice*>(out);

	bool isImageDeviceOwned = false;

	// In case the user provided a host-side image
	std::unique_ptr<ImageDeviceOwned> deviceImg_out = nullptr;
	Image* hostImg_out = nullptr;
	if (img_out == nullptr)
	{
		hostImg_out = dynamic_cast<Image*>(out);
		ASSERT_MSG(
		    hostImg_out != nullptr,
		    "The image provided is not a ImageDevice nor a Image (host)");

		deviceImg_out = std::make_unique<ImageDeviceOwned>(
		    hostImg_out->getParams(), getAuxStream());
		deviceImg_out->allocate(false);
		deviceImg_out->transferToDeviceMemory(hostImg_out, false);

		// Use owned ImageDevice
		img_out = deviceImg_out.get();
		isImageDeviceOwned = true;
	}

	ProjectionDataDevice* dat_in = nullptr;
	bool isProjDataDeviceOwned = false;

	// In case the user provided a Host-side ProjectionData
	std::unique_ptr<ProjectionDataDeviceOwned> deviceDat_in = nullptr;
	if (dat_in_const == nullptr)
	{
		auto* hostDat_in = dynamic_cast<const ProjectionData*>(in);
		ASSERT_MSG(hostDat_in != nullptr,
		           "The Projection Data provded is not a ProjectionDataDevice "
		           "nor a ProjectionData (host)");
		ASSERT_MSG(binIter != nullptr, "BinIterator undefined");

		std::vector<const BinIterator*> binIterators;
		binIterators.push_back(binIter);  // We project only one subset
		deviceDat_in = std::make_unique<ProjectionDataDeviceOwned>(
		    getScanner(), hostDat_in, binIterators);

		// Use owned ProjectionDataDevice
		dat_in = deviceDat_in.get();
		isProjDataDeviceOwned = true;
	}
	else
	{
		dat_in = const_cast<ProjectionDataDevice*>(dat_in_const);
		ASSERT_MSG(dat_in != nullptr,
		           "ProjectionDataDevice is null. Cast failed");
	}

	if (!isProjDataDeviceOwned)
	{
		std::cout << "Backprojecting current batch..." << std::endl;
		applyOnLoadedBatch<false>(dat_in, img_out);
	}
	else
	{
		// Iterate over all the batches of the current subset
		const size_t numBatches = dat_in->getBatchSetup(0).getNumBatches();
		const ImageParams& imgParams = img_out->getParams();
		for (size_t batchId = 0; batchId < numBatches; batchId++)
		{
			std::cout << "Loading batch " << batchId + 1 << "/" << numBatches
			          << "..." << std::endl;
			dat_in->loadEventLORs(0, batchId, imgParams, getAuxStream());
			deviceDat_in->allocateForProjValues(getAuxStream());
			deviceDat_in->loadProjValuesFromReference(getAuxStream());
			std::cout << "Backprojecting batch..." << std::endl;
			applyOnLoadedBatch<false>(dat_in, img_out);
		}
	}

	if (isImageDeviceOwned)
	{
		// Need to transfer the generated image back to the host
		deviceImg_out->transferToHostMemory(hostImg_out, false);
	}
}

template <bool IsForward>
void OperatorProjectorDD_GPU::applyOnLoadedBatch(ProjectionDataDevice* dat,
                                                 ImageDevice* img)
{
	setBatchSize(dat->getCurrentBatchSize());
	const auto cuScannerParams = getCUScannerParams(getScanner());
	const auto cuImageParams = getCUImageParams(img->getParams());
	const TimeOfFlightHelper* tofHelperDevicePointer =
	    getTOFHelperDevicePointer();
	const float* projPsfDevicePointer =
	    getProjPsfKernelsDevicePointer(!IsForward);

	if (projPsfDevicePointer == nullptr)
	{
		if (tofHelperDevicePointer == nullptr)
		{
			launchKernel<IsForward, false, false>(
			    dat->getProjValuesDevicePointer(), img->getDevicePointer(),
			    dat->getLorDet1PosDevicePointer(),
			    dat->getLorDet2PosDevicePointer(),
			    dat->getLorDet1OrientDevicePointer(),
			    dat->getLorDet2OrientDevicePointer(), nullptr /*No TOF*/,
			    nullptr /*No TOF*/, nullptr /*No ProjPSF*/, {} /*No ProjPSF*/,
			    cuScannerParams, cuImageParams, getBatchSize(), getGridSize(),
			    getBlockSize(), getMainStream(), isSynchronized());
		}
		else
		{
			launchKernel<IsForward, true, false>(
			    dat->getProjValuesDevicePointer(), img->getDevicePointer(),
			    dat->getLorDet1PosDevicePointer(),
			    dat->getLorDet2PosDevicePointer(),
			    dat->getLorDet1OrientDevicePointer(),
			    dat->getLorDet2OrientDevicePointer(),
			    dat->getLorTOFValueDevicePointer(), tofHelperDevicePointer,
			    nullptr /*No ProjPSF*/, {} /*No ProjPSF*/, cuScannerParams,
			    cuImageParams, getBatchSize(), getGridSize(), getBlockSize(),
			    getMainStream(), isSynchronized());
		}
	}
	else
	{
		const ProjectionPsfProperties projectionPsfProperties =
		    mp_projPsfManager->getProjectionPsfProperties();

		if (tofHelperDevicePointer == nullptr)
		{
			launchKernel<IsForward, false, true>(
			    dat->getProjValuesDevicePointer(), img->getDevicePointer(),
			    dat->getLorDet1PosDevicePointer(),
			    dat->getLorDet2PosDevicePointer(),
			    dat->getLorDet1OrientDevicePointer(),
			    dat->getLorDet2OrientDevicePointer(), nullptr /*No TOF*/,
			    nullptr /*No TOF*/, projPsfDevicePointer,
			    projectionPsfProperties, cuScannerParams, cuImageParams,
			    getBatchSize(), getGridSize(), getBlockSize(), getMainStream(),
			    isSynchronized());
		}
		else
		{
			launchKernel<IsForward, true, true>(
			    dat->getProjValuesDevicePointer(), img->getDevicePointer(),
			    dat->getLorDet1PosDevicePointer(),
			    dat->getLorDet2PosDevicePointer(),
			    dat->getLorDet1OrientDevicePointer(),
			    dat->getLorDet2OrientDevicePointer(),
			    dat->getLorTOFValueDevicePointer(), tofHelperDevicePointer,
			    projPsfDevicePointer, projectionPsfProperties, cuScannerParams,
			    cuImageParams, getBatchSize(), getGridSize(), getBlockSize(),
			    getMainStream(), isSynchronized());
		}
	}
}

template <bool IsForward, bool HasTOF, bool HasProjPsf>
void OperatorProjectorDD_GPU::launchKernel(
    float* pd_projValues, float* pd_image, const float4* pd_lorDet1Pos,
    const float4* pd_lorDet2Pos, const float4* pd_lorDet1Orient,
    const float4* pd_lorDet2Orient, const float* pd_lorTOFValue,
    const TimeOfFlightHelper* pd_tofHelper, const float* pd_projPsfKernels,
    ProjectionPsfProperties projectionPsfProperties,
    CUScannerParams scannerParams, CUImageParams imgParams, size_t batchSize,
    unsigned int gridSize, unsigned int blockSize, const hipStream_t* stream,
    bool synchronize)
{
	ASSERT_MSG(pd_projValues != nullptr && pd_lorDet1Pos != nullptr &&
	               pd_lorDet2Pos != nullptr && pd_lorDet1Orient != nullptr &&
	               pd_lorDet2Orient != nullptr,
	           "Projection space not allocated on device");
	ASSERT_MSG(pd_image != nullptr, "Image space not allocated on device");

	if (stream != nullptr)
	{
		OperatorProjectorDDCU_kernel<IsForward, HasTOF, HasProjPsf>
		    <<<gridSize, blockSize, 0, *stream>>>(
		        pd_projValues, pd_image, pd_lorDet1Pos, pd_lorDet2Pos,
		        pd_lorDet1Orient, pd_lorDet2Orient, pd_lorTOFValue,
		        pd_tofHelper, pd_projPsfKernels, projectionPsfProperties,
		        scannerParams, imgParams, batchSize);
		if (synchronize)
		{
			hipStreamSynchronize(*stream);
		}
	}
	else
	{
		OperatorProjectorDDCU_kernel<IsForward, HasTOF, HasProjPsf>
		    <<<gridSize, blockSize>>>(
		        pd_projValues, pd_image, pd_lorDet1Pos, pd_lorDet2Pos,
		        pd_lorDet1Orient, pd_lorDet2Orient, pd_lorTOFValue,
		        pd_tofHelper, pd_projPsfKernels, projectionPsfProperties,
		        scannerParams, imgParams, batchSize);
		if (synchronize)
		{
			hipDeviceSynchronize();
		}
	}
	cudaCheckError();
}
