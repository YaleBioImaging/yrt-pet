/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "yrt-pet/operators/OperatorProjectorSiddon_GPU.cuh"

#include "yrt-pet/datastruct/image/Image.hpp"
#include "yrt-pet/datastruct/image/ImageDevice.cuh"
#include "yrt-pet/datastruct/projection/ProjectionDataDevice.cuh"
#include "yrt-pet/operators/OperatorProjectorSiddon_GPUKernels.cuh"
#include "yrt-pet/utils/Assert.hpp"
#include "yrt-pet/utils/GPUUtils.cuh"

#if BUILD_PYBIND11
#include <pybind11/pybind11.h>
namespace py = pybind11;

namespace yrt
{
void py_setup_operatorprojectorsiddon_gpu(py::module& m)
{
	auto c = py::class_<OperatorProjectorSiddon_GPU, OperatorProjectorDevice>(
	    m, "OperatorProjectorSiddon_GPU");
	// c.def(py::init<const OperatorProjectorParams&>(), py::arg("projParams"));
	c.def(
	    "__init__",
	    [](const OperatorProjectorSiddon_GPU& self,
	       const OperatorProjectorParams& params)
	    { return OperatorProjectorSiddon_GPU(params, {}); },
	    py::arg("projParams"));
}
}  // namespace yrt

#endif

namespace yrt
{
OperatorProjectorSiddon_GPU::OperatorProjectorSiddon_GPU(
    const OperatorProjectorParams& projParams,
    const std::vector<Constraint*>& constraints, const hipStream_t* mainStream,
    const hipStream_t* auxStream)
    : OperatorProjectorDevice(projParams, constraints, mainStream, auxStream),
      m_numRays{projParams.numRays}
{
	initBinIteratorConstrained(projParams.projPropertyTypesExtra,
	                           projParams.numThreads);
}

std::set<ProjectionPropertyType>
    OperatorProjectorSiddon_GPU::getProjectionPropertyTypes() const
{
	std::set<ProjectionPropertyType> props{ProjectionPropertyType::LOR};
	if (m_numRays > 1)
	{
		props.insert(ProjectionPropertyType::DetOrient);
	}
	return props;
}

void OperatorProjectorSiddon_GPU::applyAOnLoadedBatch(ImageDevice& img,
                                                      ProjectionDataDevice& dat,
                                                      bool synchronize)
{
	applyOnLoadedBatch<true>(dat, img, synchronize);
}
void OperatorProjectorSiddon_GPU::applyAHOnLoadedBatch(
    ProjectionDataDevice& dat, ImageDevice& img, bool synchronize)
{
	applyOnLoadedBatch<false>(dat, img, synchronize);
}

template <bool IsForward>
void OperatorProjectorSiddon_GPU::applyOnLoadedBatch(ProjectionDataDevice& dat,
                                                     ImageDevice& img,
                                                     bool synchronize)
{
	setBatchSize(dat.getLoadedBatchSize());
	const auto cuScannerParams = getCUScannerParams(getScanner());
	const auto cuImageParams = getCUImageParams(img.getParams());
	const ProjectionPropertyManager* projPropManager =
	    getProjPropManagerDevicePointer();
	const TimeOfFlightHelper* tofHelperDevicePointer =
	    getTOFHelperDevicePointer();

	// We assume there is no Projection-space PSF to do
	if (tofHelperDevicePointer == nullptr)
	{
		OperatorProjectorSiddon_GPU::launchKernel<IsForward, false>(
		    dat.getProjValuesDevicePointer(), img.getDevicePointer(),
		    dat.getProjectionPropertiesDevicePointer(), projPropManager,
		    nullptr /*No TOF*/, cuScannerParams, cuImageParams, getBatchSize(),
		    getGridSize(), getBlockSize(), getMainStream(), synchronize);
	}
	else
	{
		OperatorProjectorSiddon_GPU::launchKernel<IsForward, true>(
		    dat.getProjValuesDevicePointer(), img.getDevicePointer(),
		    dat.getProjectionPropertiesDevicePointer(), projPropManager,
		    tofHelperDevicePointer, cuScannerParams, cuImageParams,
		    getBatchSize(), getGridSize(), getBlockSize(), getMainStream(),
		    synchronize);
	}
}

template <bool IsForward, bool HasTOF>
void OperatorProjectorSiddon_GPU::launchKernel(
    float* pd_projValues, float* pd_image, const char* pd_projProperties,
    const ProjectionPropertyManager* pd_projPropManager,
    const TimeOfFlightHelper* pd_tofHelper, CUScannerParams scannerParams,
    CUImageParams imgParams, size_t batchSize, unsigned int gridSize,
    unsigned int blockSize, const hipStream_t* stream, bool synchronize)
{
	ASSERT_MSG(pd_projValues != nullptr && pd_projPropManager != nullptr,
	           "Projection space not allocated on device");
	ASSERT_MSG(pd_image != nullptr, "Image space not allocated on device");

	if (m_numRays == 1)
	{
		if (stream != nullptr)
		{
			OperatorProjectorSiddonCU_kernel<IsForward, HasTOF, true, false>
			    <<<gridSize, blockSize, 0, *stream>>>(
			        pd_projValues, pd_image, pd_projProperties,
			        pd_projPropManager, pd_tofHelper, scannerParams, imgParams,
			        1, batchSize);
			if (synchronize)
			{
				hipStreamSynchronize(*stream);
			}
		}
		else
		{
			OperatorProjectorSiddonCU_kernel<IsForward, HasTOF, true, false>
			    <<<gridSize, blockSize>>>(pd_projValues, pd_image,
			                              pd_projProperties, pd_projPropManager,
			                              pd_tofHelper, scannerParams,
			                              imgParams, 1, batchSize);
			if (synchronize)
			{
				hipDeviceSynchronize();
			}
		}
	}
	else
	{
		if (stream != nullptr)
		{
			OperatorProjectorSiddonCU_kernel<IsForward, HasTOF, true, true>
			    <<<gridSize, blockSize, 0, *stream>>>(
			        pd_projValues, pd_image, pd_projProperties,
			        pd_projPropManager, pd_tofHelper, scannerParams, imgParams,
			        m_numRays, batchSize);
			if (synchronize)
			{
				hipStreamSynchronize(*stream);
			}
		}
		else
		{
			OperatorProjectorSiddonCU_kernel<IsForward, HasTOF, true, true>
			    <<<gridSize, blockSize>>>(pd_projValues, pd_image,
			                              pd_projProperties, pd_projPropManager,
			                              pd_tofHelper, scannerParams,
			                              imgParams, m_numRays, batchSize);
			if (synchronize)
			{
				hipDeviceSynchronize();
			}
		}
	}
	cudaCheckError();
}
}  // namespace yrt
