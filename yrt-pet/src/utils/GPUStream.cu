/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "yrt-pet/utils/GPUStream.cuh"

namespace yrt
{
GPUStream::GPUStream(unsigned int flags)
{
	hipStreamCreateWithFlags(&m_stream, flags);
}

const hipStream_t& GPUStream::getStream() const
{
	return m_stream;
}

GPUStream::~GPUStream()
{
	hipStreamDestroy(m_stream);
}
}  // namespace yrt
