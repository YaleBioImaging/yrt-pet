#include "hip/hip_runtime.h"
/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "catch.hpp"

#include "../test_utils.hpp"
#include "datastruct/image/Image.hpp"
#include "datastruct/projection/ListModeLUT.hpp"
#include "utils/ReconstructionUtils.hpp"

TEST_CASE("siddon_gpu_vs_cpu", "[siddon-gpu]")
{
	// Create Scanner
	const auto scanner = TestUtils::makeScanner();

	const size_t numDets = scanner->getTheoreticalNumDets();

	// Setup image
	constexpr int nx = 256;
	constexpr int ny = 256;
	constexpr int nz = 128;
	constexpr float ox = 3.0f;
	constexpr float oy = 10.0f;
	constexpr float oz = -15.0f;
	const float sx = scanner->scannerRadius * 2.0f / sqrt(2.0f);
	const float sy = scanner->scannerRadius * 2.0f / sqrt(2.0f) - oy * 2.0f;
	const float sz = scanner->axialFOV;
	ImageParams imgParams{nx, ny, nz, sx, sy, sz, ox, oy, oz};
	auto img = std::make_unique<ImageOwned>(imgParams);
	img->allocate();

	auto data = std::make_unique<ListModeLUTOwned>(*scanner);
	constexpr size_t numEvents = 1000;
	data->allocate(numEvents);

	for (bin_t binId = 0; binId < numEvents; binId++)
	{
		const det_id_t d1 = rand() % numDets;
		const det_id_t d2 = rand() % numDets;
		data->setDetectorIdsOfEvent(binId, d1, d2);
	}

	auto img_cpu = std::make_unique<ImageOwned>(imgParams);
	img_cpu->allocate();
	img_cpu->setValue(0.0);
	Util::backProject(*scanner, *img_cpu, *data, OperatorProjector::SIDDON,
	                  false);

	REQUIRE(img_cpu->voxelSum() > 0.0f);

	auto img_gpu = std::make_unique<ImageOwned>(imgParams);
	img_gpu->allocate();
	img_gpu->setValue(0.0);
	Util::backProject(*scanner, *img_gpu, *data, OperatorProjector::SIDDON,
	                  true);

	double rmseCpuGpu = TestUtils::getRMSE(*img_gpu, *img_cpu);

	REQUIRE(img_gpu->voxelSum() > 0.0f);

	CHECK(rmseCpuGpu < 0.000005);

	const Image& imgToFwdProj = *img_cpu;

	auto projList_cpu = std::make_unique<ProjectionListOwned>(data.get());
	projList_cpu->allocate();
	projList_cpu->clearProjections(0.0f);
	Util::forwProject(*scanner, imgToFwdProj, *projList_cpu,
	                  OperatorProjector::SIDDON, false);

	auto projList_gpu = std::make_unique<ProjectionListOwned>(data.get());
	projList_gpu->allocate();
	projList_gpu->clearProjections(0.0f);
	Util::forwProject(*scanner, imgToFwdProj, *projList_gpu,
	                  OperatorProjector::SIDDON, true);

	rmseCpuGpu = TestUtils::getRMSE(*projList_cpu, *projList_gpu);

	CHECK(rmseCpuGpu < 0.0001);
}
